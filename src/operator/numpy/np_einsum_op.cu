/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_einsum_op.cu
 * \brief GPU Implementation of numpy-compatible einsum
 */

#include "./np_einsum_op-inl.h"

namespace mxnet {
namespace op {

#if MXNET_USE_CUTENSOR == 1
template<typename U>
struct CuTensorTypeTraits;
template<>
struct CuTensorTypeTraits<double> {
  static const hipDataType cudaType = HIP_R_64F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_64F;
  typedef double ScalarType;
};
template<>
struct CuTensorTypeTraits<float> {
  static const hipDataType cudaType = HIP_R_32F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_32F;
  typedef float ScalarType;
};
template<>
struct CuTensorTypeTraits<mshadow::half::half_t> {
  static const hipDataType cudaType = HIP_R_16F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_16F;
  typedef float ScalarType;
};
using ModeType = int32_t;

// Round a value 'x' up to the next multiple of 'multiple'
size_t RoundToMultiple(size_t x, size_t multiple) {
  size_t retVal = ((x + multiple - 1) / multiple) * multiple;
  return retVal;
}
}  // namespace op
}  // namespace mxnet

namespace std {
template<>
struct hash<mxnet::op::EinsumOp> {
  size_t operator()(const mxnet::op::EinsumOp& val) {
    size_t ret = 0;
    ret = dmlc::HashCombine(ret, val.num_args);
    ret = dmlc::HashCombine(ret, val.subscripts);
    ret = dmlc::HashCombine(ret, val.optimize);
    return ret;
  }
};
}  // namespace std

namespace mxnet {
namespace op {
/*!
 * \brief The Operator used to perform einsum using cuTensor library.
 */
template<typename DType>
class CuTensorEinsum {
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
 public:
  CuTensorEinsum() {
  }
  ~CuTensorEinsum() {
  }
  void InitializeModes(std::string subscripts,
                       const mxnet::TShape& a_shape,
                       const mxnet::TShape& b_shape,
                       std::unordered_map<ModeType, int64_t>& mode_2_size,
                       std::vector<ModeType>& modes_a,
                       std::vector<ModeType>& modes_b,
                       std::vector<ModeType>& modes_c) {
    std::string equation(subscripts);
    auto end_pos = std::remove(equation.begin(), equation.end(), ' ');
    equation.erase(end_pos, equation.end());
    int comma_pos = equation.find(",");
    int arrow_pos = equation.find("->", comma_pos + 1);
    int a_begin = 0;
    int a_end = comma_pos;
    int b_begin = comma_pos + 1;
    int b_end = arrow_pos;
    int c_begin = arrow_pos + 2;
    int c_end = equation.size();

    assert((a_end - a_begin) == a_shape.ndim());
    for (int i = a_begin; i < a_end; i++) {
      mode_2_size[equation.at(i)] = a_shape[i - a_begin];
    }
    assert((b_end - b_begin) == b_shape.ndim());
    for (int i = b_begin; i < b_end; i++) {
      if (mode_2_size.find(equation.at(i)) == mode_2_size.end()) {
        mode_2_size[equation.at(i)] = b_shape[i - b_begin];
      } else {
        assert(b_shape[i - b_begin] == mode_2_size[equation.at(i)]);
      }
    }
    for (int i = a_end-1; i >= a_begin; i--) {
      modes_a.push_back(equation.at(i));
    }
    for (int i = b_end-1; i >= b_begin; i--) {
      modes_b.push_back(equation.at(i));
    }
    for (int i = c_end-1; i >= c_begin; i--) {
      modes_c.push_back(equation.at(i));
    }
  }

  void Init(std::string equation,
            const std::vector<TBlob>& inputs,
            const std::vector<TBlob>& outputs,
            const OpContext &ctx,
            bool req_write,
            size_t prev_workspace_size) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(inputs.size(), 2);
    CHECK_EQ(outputs.size(), 1);
    mxnet::TShape a_shape = inputs[0].shape_;
    mxnet::TShape b_shape = inputs[1].shape_;
    mxnet::TShape c_shape = outputs[0].shape_;

    cudaType = CuTensorTypeTraits<DType>::cudaType;
    cutensorType = CuTensorTypeTraits<DType>::cutensorType;
    // using defaul algo
    algo = HIPTENSOR_ALGO_DEFAULT;
    
    // initialize modes
    InitializeModes(equation.c_str(),
                    a_shape, b_shape,
                    mode_2_size, 
                    modes_a, modes_b, modes_c);

    std::vector<int64_t> sizes_a;
    for(auto mode : modes_a)
        sizes_a.push_back(mode_2_size[mode]);
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_a,
                                               a_shape.ndim(),
                                               sizes_a.data(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));
    std::vector<int64_t> sizes_b;
    for(auto mode : modes_b)
        sizes_b.push_back(mode_2_size[mode]);
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_b,
                                               b_shape.ndim(),
                                               sizes_b.data(),
                                               NULL, //stride
                                               cudaType, HIPTENSOR_OP_IDENTITY));
    std::vector<int64_t> sizes_c;
    for(auto mode : modes_c)
        sizes_c.push_back(mode_2_size[mode]);
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_c,
                                               c_shape.ndim(),
                                               sizes_c.data(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));

    const DType* tensor_a_ptr =  inputs[0].FlatTo2D<gpu, DType>(s).dptr_;
    const DType* tensor_b_ptr =  inputs[1].FlatTo2D<gpu, DType>(s).dptr_;
    DType* tensor_c_ptr =  outputs[0].FlatTo2D<gpu, DType>(s).dptr_;
    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_a_ptr,
                                                  &descriptor_a,
                                                  &alignment_req_a));

    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_b_ptr,
                                                  &descriptor_b,
                                                  &alignment_req_b));

    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_c_ptr,
                                                  &descriptor_c,
                                                  &alignment_req_c));

    CUTENSOR_CALL(cutensorInitContractionDescriptor(
                  &s->cutensor_handle_,
                  &descriptor_contraction,
                  &descriptor_a, modes_a.data(), alignment_req_a,
                  &descriptor_b, modes_b.data(), alignment_req_b,
                  &descriptor_c, modes_c.data(), alignment_req_c,
                  &descriptor_c, modes_c.data(), alignment_req_c,
                  cutensorType));

    CUTENSOR_CALL(cutensorInitContractionFind(&s->cutensor_handle_,
                                              &find, algo));
    previous_workspace_size = prev_workspace_size * sizeof(DType);
    CUTENSOR_CALL(cutensorContractionGetWorkspace(&s->cutensor_handle_,
                                                  &descriptor_contraction,
                                                  &find,
                                                  HIPTENSOR_WORKSPACE_MAX,
                                                  &my_workspace_size));
    total_workspace_size = previous_workspace_size + my_workspace_size;

    CUTENSOR_CALL(cutensorInitContractionPlan(&s->cutensor_handle_,
                                              &plan,
                                              &descriptor_contraction,
                                              &find,
                                              my_workspace_size));
  }

  void Compute(const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               bool req_write,
               const std::vector<TBlob> &outputs) {
    mxnet_op::Stream<gpu>* s = ctx.get_stream<gpu>();

    const TBlob &tensor_a = inputs[0];
    const TBlob &tensor_b = inputs[1];
    const TBlob &tensor_c = outputs[0];
    const DType* tensor_a_ptr =  tensor_a.FlatTo2D<gpu, DType>(s).dptr_;
    const DType* tensor_b_ptr =  tensor_b.FlatTo2D<gpu, DType>(s).dptr_;
    DType* tensor_c_ptr =  tensor_c.FlatTo2D<gpu, DType>(s).dptr_;
    
    Tensor<gpu, 1, char> global_workspace =
        ctx.requested[0].get_space_typed<gpu, 1, char>(Shape1(total_workspace_size), s);
    Tensor<gpu, 1, char> my_workspace(&global_workspace[previous_workspace_size],
                                      Shape1(my_workspace_size), s);

    CUTENSOR_CALL(hiptensorContraction(&s->cutensor_handle_,
                                      &plan,
                                      (void*) &alpha, tensor_a_ptr, tensor_b_ptr,
                                      (void*) &beta,  tensor_c_ptr, tensor_c_ptr,
                                      my_workspace.dptr_,
                                      my_workspace_size,
                                      mshadow::Stream<gpu>::GetStream(s)));
  }

  // modes
  std::unordered_map<ModeType, int64_t> mode_2_size;
  std::vector<int> modes_a;
  std::vector<int> modes_b;
  std::vector<int> modes_c;

  // descriptors
  hiptensorTensorDescriptor_t descriptor_a;
  hiptensorTensorDescriptor_t descriptor_b;
  hiptensorTensorDescriptor_t descriptor_c;
  cutensorContractionDescriptor_t descriptor_contraction;
  // aligments
  uint32_t alignment_req_a;
  uint32_t alignment_req_b;
  uint32_t alignment_req_c;

  // contraction plan and algo
  hiptensorContractionPlan_t plan;
  cutensorContractionFind_t find;
  hiptensorAlgo_t algo;

  // workspace
  size_t previous_workspace_size = 0;
  size_t my_workspace_size = 0;
  size_t total_workspace_size = 0;
  
  typename CuTensorTypeTraits<DType>::ScalarType alpha = 1;
  typename CuTensorTypeTraits<DType>::ScalarType beta = 0;

  hipDataType cudaType;
  hiptensorComputeDescriptor_t cutensorType;
};
// end CuTensorEinsum class

template<typename DType>
class EinsumOpGPU {

 public:
  EinsumOpGPU() {
  }
  ~EinsumOpGPU() {
  }

  void Init(const EinsumOp& state,
            const std::vector<TBlob>& inputs,
            const RunContext& rctx,
            bool req_write) {
    if (state.num_args == 2) {
      fwd_cutensor_ops.push_back(CuTensorEinsum<DType>());
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      for (int i = 0; i + 1 < paths_len; ++i) {
        temp_ouputs_size += state.paths[i].oshape.Size();
      }
      temp_ouputs_size_aligned = RoundToMultiple(temp_ouputs_size, dptr_alignment);
    }
  }

  void Forward(const EinsumOp& state,
               const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               const std::vector<OpReqType>& req,
               const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    bool req_write = false;
    if (state.num_args == 2) {
      fwd_cutensor_ops[0].Init(state.subscripts,
                               inputs, outputs,
                               ctx, req_write,
                               0);
      fwd_cutensor_ops[0].Compute(ctx, inputs, req_write, outputs);
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      std::vector<TBlob> operands(inputs);
      std::vector<TBlob> tmp_operands;
      std::vector<TBlob> temp_space_vec(paths_len - 1);
      Tensor<gpu, 1, DType> temp_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      size_t begin = 0;
      for (int i = 0; i < paths_len - 1; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_space_vec[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      for (int i = 0; i < paths_len; ++i) {
        bool handle_out = (i == paths_len - 1);
        tmp_operands.clear();
        // remove inds from right to left
        for (const int& p : state.paths[i].contract_inds) {
          tmp_operands.push_back(operands[p]);
          operands.erase(operands.begin() + p);
        }
        CuTensorEinsum<DType> cuTensor_einsum = CuTensorEinsum<DType>();
        cuTensor_einsum.Init(state.paths[i].einsum_str,
                             tmp_operands,
                             handle_out ? outputs : std::vector<TBlob>{temp_space_vec[i]},
                             ctx, req_write,
                             temp_ouputs_size_aligned);
        cuTensor_einsum.Compute(ctx, tmp_operands, req_write,
                                handle_out ? outputs : std::vector<TBlob>{temp_space_vec[i]});
        if (!handle_out) {
          operands.push_back(temp_space_vec[i]);
        }
      }
    }
  }

  void ComputeGradients(std::string equation,
                        const std::vector<TBlob> &inputs,
                        const std::vector<TBlob> &outputs,
                        const OpContext &ctx){
    bool req_write = true;
    int comma_pos = equation.find(",");
    int arrow_pos = equation.find("->", comma_pos + 1);
    int len_op2 = arrow_pos - comma_pos - 1;

    // gradient for first operand
    std::vector<TBlob> grad_operand1_inputs;
    std::vector<TBlob> grad_operand1_outputs;
    grad_operand1_inputs.push_back(inputs[0]);
    grad_operand1_inputs.push_back(inputs[2]);
    grad_operand1_outputs.push_back(outputs[0]);
    std::string grad_operand1_equation = equation.substr(arrow_pos + 2);
    grad_operand1_equation += ",";
    grad_operand1_equation += equation.substr(comma_pos + 1, len_op2);
    grad_operand1_equation += "->";
    grad_operand1_equation += equation.substr(0, comma_pos);
    CuTensorEinsum<DType> cuTensor_einsum1 = CuTensorEinsum<DType>();
    cuTensor_einsum1.Init(grad_operand1_equation,
                          grad_operand1_inputs,
                          grad_operand1_outputs,
                          ctx, req_write,
                          temp_ouputs_size_aligned);
    cuTensor_einsum1.Compute(ctx, grad_operand1_inputs, req_write,
                             grad_operand1_outputs);
    // gradient for second operand
    std::vector<TBlob> grad_operand2_inputs;
    std::vector<TBlob> grad_operand2_outputs;
    grad_operand2_inputs.push_back(inputs[1]);
    grad_operand2_inputs.push_back(inputs[0]);
    grad_operand2_outputs.push_back(outputs[1]);
    std::string grad_operand2_equation = equation.substr(0, comma_pos);
    grad_operand2_equation += ",";
    grad_operand2_equation += equation.substr(arrow_pos + 2);
    grad_operand2_equation += "->";
    grad_operand2_equation += equation.substr(comma_pos + 1, len_op2);

    CuTensorEinsum<DType> cuTensor_einsum2 = CuTensorEinsum<DType>();
    cuTensor_einsum2.Init(grad_operand2_equation,
                          grad_operand2_inputs,
                          grad_operand2_outputs,
                          ctx, req_write,
                          0);
    cuTensor_einsum2.Compute(ctx, grad_operand2_inputs, req_write,
                             grad_operand2_outputs);
  }

  void Backward(const EinsumOp& state,
                const OpContext &ctx,
                const std::vector<TBlob> &inputs,
                const std::vector<OpReqType>& req,
                const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    auto req_write = req[0] == kWriteTo;
    if (state.num_args == 2) {
      // inputs: out_grad, operand1, operand2
      // outputs: grad_operand1, grad_operand2
      ComputeGradients(state.subscripts,
                       inputs, outputs,
                       ctx);
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      // replay the forward process
      std::vector<std::vector<int> > op_idx(paths_len + 1);
      for (int i = 0; i <= paths_len; ++i) {
        if (i == 0) {
          op_idx[i].reserve(state.num_args);
          for (int j = 0; j < state.num_args; ++j) {
            op_idx[i].push_back(j + 1);
          }
        } else {
          op_idx[i] = op_idx[i - 1];
          // remove inds from right to left
          for (const int& p : state.paths[i - 1].contract_inds) {
            op_idx[i].erase(op_idx[i].begin() + p);
          }
          op_idx[i].push_back(-static_cast<int>(i - 1));
        }
      }
      // allocate temporary space and propagate
      std::vector<TBlob> temp_grad(paths_len - 1), temp_data(paths_len - 1);
      // outputs from forward pass, no need to re-compute, take from state
      Tensor<gpu, 1, DType> ndarray_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      size_t begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(ndarray_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_data[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // temporal grads
      Tensor<gpu, 1, DType> temp_space =
        ctx.requested[0].get_space_typed<gpu, 1, DType>(Shape1(temp_ouputs_size_aligned), s);
      begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_grad[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // go through the paths in the reversed order
      std::vector<TBlob> temp_inputs, temp_outputs;
      //std::vector<OpReqType> temp_req;
      for (int i = paths_len - 1; i >= 0; i--) {
        temp_inputs.clear();
        temp_outputs.clear();
        //temp_req.clear();
        bool handle_out = (i == paths_len - 1);
        if (handle_out) {
          // grad_out
          temp_inputs.push_back(inputs[0]);
        } else {
          temp_inputs.push_back(temp_grad[i]);
        }
        for (auto p : state.paths[i].contract_inds) {
          int idx = op_idx[i][p];
          if (idx >= 1) {
            temp_inputs.push_back(inputs[idx]);
            temp_outputs.push_back(outputs[idx - 1]);
            //temp_req.push_back(req[idx - 1]);
          } else {
            temp_inputs.push_back(temp_data[-idx]);
            temp_outputs.push_back(temp_grad[-idx]);
            //temp_req.push_back(OpReqType::kWriteTo);
          }
        }
        CHECK_EQ(temp_inputs.size(), 3U);
        CHECK_EQ(temp_outputs.size(), 2U);
        //CHECK_EQ(temp_req.size(), 2U);

        ComputeGradients(state.paths[i].einsum_str,
                         temp_inputs, temp_outputs,
                         ctx);
      }
    }
  }

  //EinsumParamGPU einsum_param;
  // cutensor ops for the forward and backward passs:
  // may not use this these if initilize descriptors at each FWD/BWD
  std::vector<CuTensorEinsum<DType>> fwd_cutensor_ops;
  std::vector<CuTensorEinsum<DType>> bwd_cutensor_ops;

  size_t temp_ouputs_size = 0;
  const size_t dptr_alignment = 512;
  size_t temp_ouputs_size_aligned = 0;
};

typedef ParamOpSign<EinsumOp> EinsumSignature;
template<typename DType>
static EinsumOpGPU<DType>& GetEinsumOpGPU(const EinsumOp& state,
                                          const std::vector<TBlob>& inputs,
                                          const std::vector<TBlob>& outputs,
                                          const RunContext& rctx,
                                          bool req_write) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local std::unordered_map<EinsumSignature,
                                         std::shared_ptr<EinsumOpGPU<DType> >,
                                         OpHash> ops;
#else
  static MX_THREAD_LOCAL std::unordered_map<EinsumSignature,
                                            std::shared_ptr<EinsumOpGPU<DType> >,
                                            OpHash> ops;
#endif
  EinsumSignature key(state);
  size_t ndim = 0;
  mxnet::ShapeVector in_shape(inputs.size());
  mxnet::ShapeVector out_shape(1, outputs[0].shape_);
  for (size_t i = 0; i < in_shape.size(); i++)
    in_shape[i] = inputs[i].shape_;
  for (auto &s : in_shape)
    ndim += s.ndim();
  for (auto &s : out_shape)
    ndim += s.ndim();
  key.Reserve(ndim + // for in and out shapes
              1 + // for dev_id
              1 ); // for req_write
  key.AddSign(in_shape);
  key.AddSign(out_shape);
  key.AddSign(rctx.ctx.dev_id);
  key.AddSign(req_write ? 1 : 0);

  auto it = ops.find(key);
  if (it == ops.end()) {
    std::shared_ptr<EinsumOpGPU<DType>> op(new EinsumOpGPU<DType>());
    auto ins_ret = ops.insert(std::pair<EinsumSignature, std::shared_ptr<EinsumOpGPU<DType>>>(
                              key, op));
    CHECK(ins_ret.second);
    it = ins_ret.first;
    it->second->Init(state,
                     inputs,
                     rctx, req_write);
  }
  return *it->second;
}
#endif

inline void NumpyEinsumForwardGpu(const OpStatePtr& state_ptr,
                                  const OpContext& ctx,
                                  const std::vector<TBlob>& inputs,
                                  const std::vector<OpReqType>& req,
                                  const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = false;
  if (state.num_args <= 1) {
    NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    std::vector<Step>& paths = state.paths;
    std::vector<std::vector<int> > pos;
    std::string string_repr;
    paths = einsum_path(state.subscripts, inputs, true, ctx.run_ctx, &pos, &string_repr);
    //EinsumParamGPU param(state.num_args, state.subscripts);
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, inputs, outputs,
           ctx.run_ctx, req_write);
      //EinsumOpGPU<DType> op = EinsumOpGPU<DType>();
      //op.Init(state, inputs, ctx.run_ctx, req_write);
      state.tempspace.reset<NDArray>(new NDArray(TShape(Shape1(op.temp_ouputs_size)),
                                               ctx.run_ctx.ctx,
                                               false,
                                               outputs[0].type_flag_));
      op.Forward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

inline void NumpyEinsumBackwardGpu(const OpStatePtr& state_ptr,
                                   const OpContext& ctx,
                                   const std::vector<TBlob>& inputs,
                                   const std::vector<OpReqType>& req,
                                   const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  const EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = req[0] == kWriteTo;
  if (state.num_args <= 1) {
    NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      std::vector<TBlob> inputs_no_grad;
      for (int i = 1; i < inputs.size(); ++i) {
        inputs_no_grad.push_back(inputs[i]);
      }
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, inputs_no_grad, outputs,
           ctx.run_ctx, req_write);
      op.Backward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

NNVM_REGISTER_OP(_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumForwardGpu);

NNVM_REGISTER_OP(_backward_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumBackwardGpu);

}  // namespace op
}  // namespace mxnet
