/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_einsum_op.cu
 * \brief GPU Implementation of numpy-compatible einsum
 */

#include "./np_einsum_op-inl.h"

namespace mxnet {
namespace op {

#if MXNET_USE_CUTENSOR == 1
template<typename U>
struct CuTensorTypeTraits;
template<>
struct CuTensorTypeTraits<double> {
  static const hipDataType cudaType = HIP_R_64F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_64F;
  typedef double ScalarType;
};
template<>
struct CuTensorTypeTraits<float> {
  static const hipDataType cudaType = HIP_R_32F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_32F;
  typedef float ScalarType;
};
template<>
struct CuTensorTypeTraits<mshadow::half::half_t> {
  static const hipDataType cudaType = HIP_R_16F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_16F;
  typedef float ScalarType;
};
using ModeType = int32_t;

// Round a value 'x' up to the next multiple of 'multiple'
size_t RoundToMultiple(size_t x, size_t multiple) {
  size_t retVal = ((x + multiple - 1) / multiple) * multiple;
  return retVal;
}
}  // namespace op
}  // namespace mxnet

namespace std {
template<>
struct hash<mxnet::op::EinsumOp> {
  size_t operator()(const mxnet::op::EinsumOp& val) {
    size_t ret = 0;
    ret = dmlc::HashCombine(ret, val.num_args);
    ret = dmlc::HashCombine(ret, val.subscripts);
    ret = dmlc::HashCombine(ret, val.optimize);
    return ret;
  }
};
}  // namespace std

namespace mxnet {
namespace op {

template<typename ComputeType,
         typename IntType, int kMaxNumModes_>
struct Einsum
{
    Einsum(const std::string &equation,
           const mxnet::TShape &A_shape,
           const mxnet::TShape &B_shape) :
        numModesA_(A_shape.ndim()),
        numModesB_(B_shape.ndim()),
        numModesC_(0),
        isInitialized_(false)
    {
        const auto arrow_pos = equation.find("->");
        const auto comma_pos = equation.find(",");
        const auto dots = equation.find("...");
        const bool isBroadcast = (dots != std::string::npos);
        const bool isImplicit = (arrow_pos == std::string::npos);
        if (isBroadcast) // TODO
        {
            return;
        }
        const bool usesB = (comma_pos != std::string::npos);

        size_t a_start = 0;
        size_t a_end = isImplicit ? ((comma_pos == std::string::npos) ? equation.size() : comma_pos) : 
                                    ((comma_pos == std::string::npos) ? arrow_pos : comma_pos);
        size_t b_start = usesB ? comma_pos + 1 : 0;
        size_t b_end   = usesB ? (isImplicit ? equation.size() : arrow_pos) : 0;
        size_t c_start = isImplicit ? equation.size() : arrow_pos + 2;
        size_t c_end = equation.size();


        char modeA[kMaxNumModes_ + 2];
        uint32_t numModesA = 0;
        for (int i = a_start; i < a_end && numModesA < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeA[numModesA++] = equation.at(i);
            }
        }

        char modeB[kMaxNumModes_ + 2];
        uint32_t numModesB = 0;
        for (int i = b_start; i < b_end && numModesB < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeB[numModesB++] = equation.at(i);
            }
        }

        char modeC[kMaxNumModes_ + 2];
        uint32_t numModesC = 0;
        for (int i = c_start; i < c_end && numModesC < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeC[numModesC++] = equation.at(i);
            }
        }

        if ((numModesA != numModesA_) || (numModesB != numModesB_))
        {
            // substring size and shape don't match
            return;
        }
        if (numModesA_ > kMaxNumModes_ || numModesB_ > kMaxNumModes_)
        {
            // too many modes
            return;
        }

        /**
         * Copy all modes from modeA to modeC if they don't appear in modeB
         */
        auto copyModesIf = [](const char* modeA, uint32_t numModesA,
                const char* modeB, uint32_t numModesB,
                char* modeC, uint32_t &numModesC)
        {
            for (uint32_t i = 0; i < numModesA; i++)
            {
                auto mode = modeA[i];
                bool found = false;
                for(uint32_t j=0; j < numModesB; ++j){
                    if(mode == modeB[j])
                    {
                        found = true;
                        break;
                    }
                }

                if (!found) // is non-contracted mode
                {
                    modeC[numModesC++] = mode;
                    if (numModesC > kMaxNumModes_)
                    {
                        // too many modes
                        return false;
                    }
                }
            }
            return true;
        };


        std::array<char, kMaxNumModes_+1> implicitModeC;
        char* redirectModeC;
        if (isImplicit)
        {
            // we have to copy all non-contracted modes from A over to C
            if (copyModesIf(modeA, numModesA_, modeB, numModesB_, implicitModeC.data(), numModesC_) == false)
            {
                return;
            }
            // we have to copy all non-contracted modes from B over to C
            if (copyModesIf(modeB, numModesB_, modeA, numModesA_, implicitModeC.data(), numModesC_) == false)
            {
                return;
            }
            std::sort(implicitModeC.begin(), std::next(implicitModeC.begin(), numModesC_)); // modes are sorted w.r.t. lexical order
            implicitModeC[numModesC_] = '\0';
            redirectModeC = implicitModeC.data();
        }
        else
        {
            redirectModeC = modeC;
            numModesC_ = numModesC;
        }

        for (uint32_t i = 0; i < numModesA_; i++)
        {
            modesA_[i] = modeA[numModesA_ - i - 1];
            extentA_[i] = A_shape[numModesA_ - i - 1];
        }

        for (uint32_t i = 0; i < numModesB_; i++)
        {
            modesB_[i] = modeB[numModesB_ - i - 1];
            extentB_[i] = B_shape[numModesB_ - i - 1];
        }

        for (uint32_t i = 0; i < numModesC_; i++)
        {
            const auto mode = redirectModeC[numModesC_ - i - 1];
            modesC_[i] = mode;
            bool found = false;
            for (uint32_t j=0; j < numModesA_; ++j)
            {
                if (modesA_[j] == mode)
                {
                    extentC_[i] = extentA_[j];
                    found = true;
                    break;
                }
            }
            for (uint32_t j=0; !found && j < numModesB_; ++j)
            {
                if (modesB_[j] == mode)
                {
                    extentC_[i] = extentB_[j];
                    break;
                }
            }
        }

        isInitialized_ = true;
    }

    size_t getWorksize() const { return kWorksize_; }

    std::vector<IntType> getOutputShape() const
    {
        if (!isInitialized_) return {};
        std::vector<IntType> extentC(numModesC_);
        for (int i=0; i < numModesC_; ++i)
        {
            extentC[i] = extentC_.at(numModesC_ - i - 1);
        }

        return extentC;
    }

    /**
     * Computes the einsum call A,B->C
     *
     * \param[in] A_raw device pointer of A
     * \param[in] B_raw device pointer of B
     * \param[out] C_raw device pointer of C
     * \param[out] wor_raw device pointer to the scratchpad memory
     * Dispatch to contraction
     */
    bool execute(const hiptensorHandle_t *handle,
                 const void* A_raw,
                 const void* B_raw,
                 void* C_raw,
                 void *work_raw, hipStream_t stream) const
    {
        if (!isInitialized_) return false;

        hipDataType cudaType = CuTensorTypeTraits<ComputeType>::cudaType;
        hiptensorComputeDescriptor_t computeType = CuTensorTypeTraits<ComputeType>::cutensorType;

        hiptensorTensorDescriptor_t descA;
        CUTENSOR_CALL(hiptensorInitTensorDescriptor(handle,
                    &descA,
                    numModesA_,
                    extentA_.data(),
                    NULL /* = stride */,
                    cudaType, HIPTENSOR_OP_IDENTITY));

        hiptensorTensorDescriptor_t descC;
        CUTENSOR_CALL(hiptensorInitTensorDescriptor(handle,
                    &descC,
                    numModesC_,
                    extentC_.data(),
                    NULL /* = stride*/,
                    cudaType, HIPTENSOR_OP_IDENTITY));

        uint32_t alignmentRequirementA;
        CUTENSOR_CALL(cutensorGetAlignmentRequirement(handle,
                    A_raw, &descA, &alignmentRequirementA));

        uint32_t alignmentRequirementC;
        CUTENSOR_CALL(cutensorGetAlignmentRequirement(handle,
                    C_raw, &descC, &alignmentRequirementC));


        hiptensorTensorDescriptor_t descB;
        uint32_t alignmentRequirementB;
        if (numModesB_ > 0)
        {
            // dispatch to contraction
            CUTENSOR_CALL(hiptensorInitTensorDescriptor(handle,
                        &descB,
                        numModesB_,
                        extentB_.data(),
                        NULL /* = stride*/,
                        cudaType, HIPTENSOR_OP_IDENTITY));

            CUTENSOR_CALL(cutensorGetAlignmentRequirement(handle,
                        B_raw, &descB, &alignmentRequirementB));

            cutensorContractionDescriptor_t desc;
            CUTENSOR_CALL(cutensorInitContractionDescriptor(handle, &desc,
                        &descA, modesA_.data(), alignmentRequirementA,
                        &descB, modesB_.data(), alignmentRequirementB,
                        &descC, modesC_.data(), alignmentRequirementC,
                        &descC, modesC_.data(), alignmentRequirementC,
                        computeType));

            hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
            cutensorContractionFind_t find;
            CUTENSOR_CALL(cutensorInitContractionFind( 
                        handle, &find, 
                        algo));

            hiptensorContractionPlan_t plan;
            CUTENSOR_CALL(cutensorInitContractionPlan(handle,
                        &plan, &desc, &find, kWorksize_));

            typename CuTensorTypeTraits<ComputeType>::ScalarType alpha = 1;
            typename CuTensorTypeTraits<ComputeType>::ScalarType beta = 0;

            CUTENSOR_CALL(hiptensorContraction(handle, &plan,
                        (void*) &alpha, A_raw, B_raw,
                        (void*) &beta,  C_raw, C_raw,
                        work_raw, kWorksize_, stream));
        }
        else
        {
            // dispatch to reduction
            typename CuTensorTypeTraits<ComputeType>::ScalarType alpha = 1;
            typename CuTensorTypeTraits<ComputeType>::ScalarType beta = 0;
            CUTENSOR_CALL(hiptensorReduction(handle,
                        (const void*)&alpha, A_raw, &descA, modesA_.data(),
                        (const void*)&beta,  A_raw, &descC, modesC_.data(), // beta == 0 => will not be used
                        C_raw, &descC, modesC_.data(),
                        HIPTENSOR_OP_ADD, computeType, work_raw, kWorksize_, stream));
        }
        return true;
    }

    bool isInitialized() const { return isInitialized_; }

    const int64_t* getExtentsA() const { return extentA_.data(); }
    const int64_t* getExtentsB() const { return extentB_.data(); }
    const int64_t* getExtentsC() const { return extentC_.data(); }

    const int* getModesA() const { return modesA_.data(); }
    const int* getModesB() const { return modesB_.data(); }
    const int* getModesC() const { return modesC_.data(); }

    private:
    static const size_t kWorksize_ = 1024ULL * 1024ULL * 8ULL * 128ULL;
    uint32_t numModesA_;
    uint32_t numModesB_;
    uint32_t numModesC_;
    bool isInitialized_;
    std::array<int, kMaxNumModes_> modesA_;
    std::array<int, kMaxNumModes_> modesB_;
    std::array<int, kMaxNumModes_> modesC_;
    std::array<int64_t, kMaxNumModes_> extentA_;
    std::array<int64_t, kMaxNumModes_> extentB_;
    std::array<int64_t, kMaxNumModes_> extentC_;
};

/*!
 * \brief The Operator used to perform einsum using cuTensor library.
 */
template<typename DType>
class CuTensorEinsum {
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  static_assert(CUTENSOR_MAJOR >= 1 && CUTENSOR_MINOR >= 2 && CUTENSOR_PATCH >= 0, "minimal cuTENSOR 1.2.0 is required.");
 public:
  CuTensorEinsum() {
  }
  ~CuTensorEinsum() {
  }

  void Init(const std::string &equation,
            const std::vector<TBlob>& inputs,
            const std::vector<TBlob>& outputs,
            const OpContext& ctx,
            bool req_write,
            size_t prev_workspace_size) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(inputs.size(), 2);
    CHECK_EQ(outputs.size(), 1);
    mxnet::TShape a_shape = inputs[0].shape_;
    mxnet::TShape b_shape = inputs[1].shape_;
    mxnet::TShape c_shape = outputs[0].shape_;

    constexpr hipDataType cudaType = CuTensorTypeTraits<DType>::cudaType;
    constexpr hiptensorComputeDescriptor_t cutensorType = CuTensorTypeTraits<DType>::cutensorType;
    constexpr hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    Einsum<DType, int, kMaxTensorRank> myEinsum(equation, a_shape, b_shape);
    if (!myEinsum.isInitialized()) {
        CUTENSOR_CALL(HIPTENSOR_STATUS_NOT_SUPPORTED);
    }

    hiptensorTensorDescriptor_t descriptor_a;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_a,
                                               a_shape.ndim(),
                                               myEinsum.getExtentsA(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));
    hiptensorTensorDescriptor_t descriptor_b;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_b,
                                               b_shape.ndim(),
                                               myEinsum.getExtentsB(),
                                               NULL, //stride
                                               cudaType, HIPTENSOR_OP_IDENTITY));
    hiptensorTensorDescriptor_t descriptor_c;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_c,
                                               c_shape.ndim(),
                                               myEinsum.getExtentsC(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));

    const DType* tensor_a_ptr =  inputs[0].FlatTo2D<gpu, DType>(s).dptr_;
    const DType* tensor_b_ptr =  inputs[1].FlatTo2D<gpu, DType>(s).dptr_;
    DType* tensor_c_ptr =  outputs[0].FlatTo2D<gpu, DType>(s).dptr_;
    uint32_t alignment_req_a;
    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_a_ptr,
                                                  &descriptor_a,
                                                  &alignment_req_a));

    uint32_t alignment_req_b;
    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_b_ptr,
                                                  &descriptor_b,
                                                  &alignment_req_b));

    uint32_t alignment_req_c;
    CUTENSOR_CALL(cutensorGetAlignmentRequirement(&s->cutensor_handle_,
                                                  tensor_c_ptr,
                                                  &descriptor_c,
                                                  &alignment_req_c));

    CUTENSOR_CALL(cutensorInitContractionDescriptor(
                  &s->cutensor_handle_,
                  &descriptor_contraction,
                  &descriptor_a, myEinsum.getModesA(), alignment_req_a,
                  &descriptor_b, myEinsum.getModesB(), alignment_req_b,
                  &descriptor_c, myEinsum.getModesC(), alignment_req_c,
                  &descriptor_c, myEinsum.getModesC(), alignment_req_c,
                  cutensorType));

    CUTENSOR_CALL(cutensorInitContractionFind(&s->cutensor_handle_,
                                              &find, algo));

    const hiptensorAutotuneMode_t autotuneMode = HIPTENSOR_AUTOTUNE_MODE_INCREMENTAL;
    CUTENSOR_CALL(cutensorContractionFindSetAttribute(
        &s->cutensor_handle_,
        &find,
        CUTENSOR_CONTRACTION_FIND_AUTOTUNE_MODE,
        &autotuneMode,
        sizeof(hiptensorAutotuneMode_t)));

    const uint32_t incCount = 5;
    CUTENSOR_CALL(cutensorContractionFindSetAttribute(
        &s->cutensor_handle_,
        &find,
        CUTENSOR_CONTRACTION_FIND_INCREMENTAL_COUNT,
        &incCount,
        sizeof(uint32_t)));

    previous_workspace_size = prev_workspace_size * sizeof(DType);
    CUTENSOR_CALL(cutensorContractionGetWorkspace(&s->cutensor_handle_,
                                                  &descriptor_contraction,
                                                  &find,
                                                  HIPTENSOR_WORKSPACE_MAX,
                                                  &my_workspace_size));
    total_workspace_size = previous_workspace_size + my_workspace_size;
  }

  void Compute(const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               bool req_write,
               const std::vector<TBlob> &outputs) {
    mxnet_op::Stream<gpu>* s = ctx.get_stream<gpu>();

    CUTENSOR_CALL(cutensorInitContractionPlan(&s->cutensor_handle_,
                                              &plan,
                                              &descriptor_contraction,
                                              &find,
                                              my_workspace_size));

    const TBlob &tensor_a = inputs[0];
    const TBlob &tensor_b = inputs[1];
    const TBlob &tensor_c = outputs[0];
    const DType* tensor_a_ptr =  tensor_a.FlatTo2D<gpu, DType>(s).dptr_;
    const DType* tensor_b_ptr =  tensor_b.FlatTo2D<gpu, DType>(s).dptr_;
    DType* tensor_c_ptr =  tensor_c.FlatTo2D<gpu, DType>(s).dptr_;
    
    Tensor<gpu, 1, char> global_workspace =
        ctx.requested[0].get_space_typed<gpu, 1, char>(Shape1(total_workspace_size), s);
    Tensor<gpu, 1, char> my_workspace(&global_workspace[previous_workspace_size],
                                      Shape1(my_workspace_size), s);

    CUTENSOR_CALL(hiptensorContraction(&s->cutensor_handle_,
                                      &plan,
                                      (void*) &alpha, tensor_a_ptr, tensor_b_ptr,
                                      (void*) &beta,  tensor_c_ptr, tensor_c_ptr,
                                      my_workspace.dptr_,
                                      my_workspace_size,
                                      mshadow::Stream<gpu>::GetStream(s)));
  }

  static const uint32_t kMaxTensorRank = 12; // maximal tensor rank that is supported by cuTENSOR

  cutensorContractionDescriptor_t descriptor_contraction; // encodes the strucutre of the contraction
  hiptensorContractionPlan_t plan; // encodes the execution plan
  cutensorContractionFind_t find; // limits the search space (of viable candidates/implementations)

  // workspace
  size_t previous_workspace_size = 0;
  size_t my_workspace_size = 0;
  size_t total_workspace_size = 0;
  
  typename CuTensorTypeTraits<DType>::ScalarType alpha = 1;
  typename CuTensorTypeTraits<DType>::ScalarType beta = 0;
};
// end CuTensorEinsum class

template<typename DType>
class EinsumOpGPU {

 public:
  EinsumOpGPU() {
  }
  ~EinsumOpGPU() {
  }

  void Init(const EinsumOp& state,
            const std::vector<TBlob>& inputs,
            const std::vector<TBlob>& outputs,
            const OpContext& ctx,
            bool req_write) {
    if (state.num_args == 2) {
      fwd_cutensor_ops.push_back(CuTensorEinsum<DType>());
      fwd_cutensor_ops[0].Init(state.subscripts,
                               inputs, outputs,
                               ctx, req_write,
                               0);
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      for (int i = 0; i + 1 < paths_len; ++i) {
        temp_ouputs_size += state.paths[i].oshape.Size();
      }
      temp_ouputs_size_aligned = RoundToMultiple(temp_ouputs_size, dptr_alignment);
    }
  }

  void Forward(const EinsumOp& state,
               const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               const std::vector<OpReqType>& req,
               const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    bool req_write = false;
    if (state.num_args == 2) {
      //fwd_cutensor_ops[0].Init(state.subscripts,
      //                         inputs, outputs,
      //                         ctx, req_write,
      //                         0);
      fwd_cutensor_ops[0].Compute(ctx, inputs, req_write, outputs);
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      std::vector<TBlob> operands(inputs);
      std::vector<TBlob> tmp_operands;
      std::vector<TBlob> temp_space_vec(paths_len - 1);
      Tensor<gpu, 1, DType> temp_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      size_t begin = 0;
      for (int i = 0; i < paths_len - 1; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_space_vec[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      for (int i = 0; i < paths_len; ++i) {
        bool handle_out = (i == paths_len - 1);
        tmp_operands.clear();
        // remove inds from right to left
        for (const int& p : state.paths[i].contract_inds) {
          tmp_operands.push_back(operands[p]);
          operands.erase(operands.begin() + p);
        }
        CuTensorEinsum<DType> cuTensor_einsum = CuTensorEinsum<DType>();
        cuTensor_einsum.Init(state.paths[i].einsum_str,
                             tmp_operands,
                             handle_out ? outputs : std::vector<TBlob>{temp_space_vec[i]},
                             ctx, req_write,
                             temp_ouputs_size_aligned);
        cuTensor_einsum.Compute(ctx, tmp_operands, req_write,
                                handle_out ? outputs : std::vector<TBlob>{temp_space_vec[i]});
        if (!handle_out) {
          operands.push_back(temp_space_vec[i]);
        }
      }
    }
  }

  void ComputeGradients(const std::string &equation,
                        const std::vector<TBlob> &inputs,
                        const std::vector<TBlob> &outputs,
                        const OpContext &ctx){
    bool req_write = true;
    int comma_pos = equation.find(",");
    int arrow_pos = equation.find("->", comma_pos + 1);
    int len_op2 = arrow_pos - comma_pos - 1;

    // gradient for first operand
    std::vector<TBlob> grad_operand1_inputs;
    std::vector<TBlob> grad_operand1_outputs;
    grad_operand1_inputs.push_back(inputs[0]);
    grad_operand1_inputs.push_back(inputs[2]);
    grad_operand1_outputs.push_back(outputs[0]);
    std::string grad_operand1_equation = equation.substr(arrow_pos + 2);
    grad_operand1_equation += ",";
    grad_operand1_equation += equation.substr(comma_pos + 1, len_op2);
    grad_operand1_equation += "->";
    grad_operand1_equation += equation.substr(0, comma_pos);
    CuTensorEinsum<DType> cuTensor_einsum1 = CuTensorEinsum<DType>();
    cuTensor_einsum1.Init(grad_operand1_equation,
                          grad_operand1_inputs,
                          grad_operand1_outputs,
                          ctx, req_write,
                          temp_ouputs_size_aligned);
    cuTensor_einsum1.Compute(ctx, grad_operand1_inputs, req_write,
                             grad_operand1_outputs);
    // gradient for second operand
    std::vector<TBlob> grad_operand2_inputs;
    std::vector<TBlob> grad_operand2_outputs;
    grad_operand2_inputs.push_back(inputs[1]);
    grad_operand2_inputs.push_back(inputs[0]);
    grad_operand2_outputs.push_back(outputs[1]);
    std::string grad_operand2_equation = equation.substr(0, comma_pos);
    grad_operand2_equation += ",";
    grad_operand2_equation += equation.substr(arrow_pos + 2);
    grad_operand2_equation += "->";
    grad_operand2_equation += equation.substr(comma_pos + 1, len_op2);

    CuTensorEinsum<DType> cuTensor_einsum2 = CuTensorEinsum<DType>();
    cuTensor_einsum2.Init(grad_operand2_equation,
                          grad_operand2_inputs,
                          grad_operand2_outputs,
                          ctx, req_write,
                          0);
    cuTensor_einsum2.Compute(ctx, grad_operand2_inputs, req_write,
                             grad_operand2_outputs);
  }

  void Backward(const EinsumOp& state,
                const OpContext &ctx,
                const std::vector<TBlob> &inputs,
                const std::vector<OpReqType>& req,
                const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    auto req_write = req[0] == kWriteTo;
    if (state.num_args == 2) {
      // inputs: out_grad, operand1, operand2
      // outputs: grad_operand1, grad_operand2
      ComputeGradients(state.subscripts,
                       inputs, outputs,
                       ctx);
    } else {
      // more than 2 operands, compute optimal path
      int paths_len = state.paths.size();
      // replay the forward process
      std::vector<std::vector<int> > op_idx(paths_len + 1);
      for (int i = 0; i <= paths_len; ++i) {
        if (i == 0) {
          op_idx[i].reserve(state.num_args);
          for (int j = 0; j < state.num_args; ++j) {
            op_idx[i].push_back(j + 1);
          }
        } else {
          op_idx[i] = op_idx[i - 1];
          // remove inds from right to left
          for (const int& p : state.paths[i - 1].contract_inds) {
            op_idx[i].erase(op_idx[i].begin() + p);
          }
          op_idx[i].push_back(-static_cast<int>(i - 1));
        }
      }
      // allocate temporary space and propagate
      std::vector<TBlob> temp_grad(paths_len - 1), temp_data(paths_len - 1);
      // outputs from forward pass, no need to re-compute, take from state
      Tensor<gpu, 1, DType> ndarray_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      size_t begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(ndarray_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_data[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // temporal grads
      Tensor<gpu, 1, DType> temp_space =
        ctx.requested[0].get_space_typed<gpu, 1, DType>(Shape1(temp_ouputs_size_aligned), s);
      begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_grad[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // go through the paths in the reversed order
      std::vector<TBlob> temp_inputs, temp_outputs;
      //std::vector<OpReqType> temp_req;
      for (int i = paths_len - 1; i >= 0; i--) {
        temp_inputs.clear();
        temp_outputs.clear();
        //temp_req.clear();
        bool handle_out = (i == paths_len - 1);
        if (handle_out) {
          // grad_out
          temp_inputs.push_back(inputs[0]);
        } else {
          temp_inputs.push_back(temp_grad[i]);
        }
        for (auto p : state.paths[i].contract_inds) {
          int idx = op_idx[i][p];
          if (idx >= 1) {
            temp_inputs.push_back(inputs[idx]);
            temp_outputs.push_back(outputs[idx - 1]);
            //temp_req.push_back(req[idx - 1]);
          } else {
            temp_inputs.push_back(temp_data[-idx]);
            temp_outputs.push_back(temp_grad[-idx]);
            //temp_req.push_back(OpReqType::kWriteTo);
          }
        }
        CHECK_EQ(temp_inputs.size(), 3U);
        CHECK_EQ(temp_outputs.size(), 2U);
        //CHECK_EQ(temp_req.size(), 2U);

        ComputeGradients(state.paths[i].einsum_str,
                         temp_inputs, temp_outputs,
                         ctx);
      }
    }
  }

  //EinsumParamGPU einsum_param;
  // cutensor ops for the forward and backward passs:
  // may not use this these if initilize descriptors at each FWD/BWD
  std::vector<CuTensorEinsum<DType>> fwd_cutensor_ops;
  std::vector<CuTensorEinsum<DType>> bwd_cutensor_ops;

  size_t temp_ouputs_size = 0;
  const size_t dptr_alignment = 512;
  size_t temp_ouputs_size_aligned = 0;
};

typedef ParamOpSign<EinsumOp> EinsumSignature;
template<typename DType>
static EinsumOpGPU<DType>& GetEinsumOpGPU(const EinsumOp& state,
                                          const std::vector<TBlob>& inputs,
                                          const std::vector<TBlob>& outputs,
                                          const OpContext& ctx,
                                          bool req_write) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local std::unordered_map<EinsumSignature,
                                         std::shared_ptr<EinsumOpGPU<DType> >,
                                         OpHash> ops;
#else
  static MX_THREAD_LOCAL std::unordered_map<EinsumSignature,
                                            std::shared_ptr<EinsumOpGPU<DType> >,
                                            OpHash> ops;
#endif
  EinsumSignature key(state);
  size_t ndim = 0;
  mxnet::ShapeVector in_shape(inputs.size());
  mxnet::ShapeVector out_shape(1, outputs[0].shape_);
  for (size_t i = 0; i < in_shape.size(); i++)
    in_shape[i] = inputs[i].shape_;
  for (auto &s : in_shape)
    ndim += s.ndim();
  for (auto &s : out_shape)
    ndim += s.ndim();
  key.Reserve(ndim + // for in and out shapes
              1 + // for dev_id
              1 ); // for req_write
  key.AddSign(in_shape);
  key.AddSign(out_shape);
  key.AddSign(ctx.run_ctx.ctx.dev_id);
  key.AddSign(req_write ? 1 : 0);

  auto it = ops.find(key);
  if (it == ops.end()) {
    std::shared_ptr<EinsumOpGPU<DType>> op(new EinsumOpGPU<DType>());
    auto ins_ret = ops.insert(std::pair<EinsumSignature, std::shared_ptr<EinsumOpGPU<DType>>>(
                              key, op));
    CHECK(ins_ret.second);
    it = ins_ret.first;
    it->second->Init(state,
                     inputs, outputs,
                     ctx, req_write);
  }
  return *it->second;
}
#endif

inline void NumpyEinsumForwardGpu(const OpStatePtr& state_ptr,
                                  const OpContext& ctx,
                                  const std::vector<TBlob>& inputs,
                                  const std::vector<OpReqType>& req,
                                  const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = false;
  if (state.num_args <= 1) {
    NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    if (state.num_args > 2) {
      std::vector<Step>& paths = state.paths;
      std::vector<std::vector<int> > pos;
      std::string string_repr;
      paths = einsum_path(state.subscripts, inputs, true, ctx.run_ctx, &pos, &string_repr);
    }
    //EinsumParamGPU param(state.num_args, state.subscripts);
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, inputs, outputs,
           ctx, req_write);
      //EinsumOpGPU<DType> op = EinsumOpGPU<DType>();
      //op.Init(state, inputs, ctx.run_ctx, req_write);
      state.tempspace.reset<NDArray>(new NDArray(TShape(Shape1(op.temp_ouputs_size)),
                                               ctx.run_ctx.ctx,
                                               false,
                                               outputs[0].type_flag_));
      op.Forward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

inline void NumpyEinsumBackwardGpu(const OpStatePtr& state_ptr,
                                   const OpContext& ctx,
                                   const std::vector<TBlob>& inputs,
                                   const std::vector<OpReqType>& req,
                                   const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  const EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = req[0] == kWriteTo;
  if (state.num_args <= 1) {
    NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      std::vector<TBlob> inputs_no_grad;
      std::vector<TBlob> out_grad;
      for (int i = 1; i < inputs.size(); ++i) {
        inputs_no_grad.push_back(inputs[i]);
      }
      out_grad.push_back(inputs[0]);
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, inputs_no_grad, out_grad,
           ctx, req_write);
      op.Backward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

NNVM_REGISTER_OP(_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumForwardGpu);

NNVM_REGISTER_OP(_backward_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumBackwardGpu);

}  // namespace op
}  // namespace mxnet
