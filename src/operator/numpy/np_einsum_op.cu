/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file np_einsum_op.cu
 * \brief GPU Implementation of numpy-compatible einsum
 */

#include "./np_einsum_op-inl.h"

namespace mxnet {
namespace op {

#if MXNET_USE_CUTENSOR == 1
template<typename U>
struct CuTensorTypeTraits;
template<>
struct CuTensorTypeTraits<double> {
  static const hipDataType cudaType = HIP_R_64F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_64F;
  typedef double ScalarType;
};
template<>
struct CuTensorTypeTraits<float> {
  static const hipDataType cudaType = HIP_R_32F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_32F;
  typedef float ScalarType;
};
template<>
struct CuTensorTypeTraits<mshadow::half::half_t> {
  static const hipDataType cudaType = HIP_R_16F;
  static const hiptensorComputeDescriptor_t cutensorType = CUTENSOR_R_MIN_16F;
  typedef float ScalarType;
};
using ModeType = int32_t;

// Round a value 'x' up to the next multiple of 'multiple'
size_t RoundToMultiple(size_t x, size_t multiple) {
  size_t retVal = ((x + multiple - 1) / multiple) * multiple;
  return retVal;
}
}  // namespace op
}  // namespace mxnet

namespace std {
template<>
struct hash<mxnet::op::EinsumOp> {
  size_t operator()(const mxnet::op::EinsumOp& val) {
    size_t ret = 0;
    ret = dmlc::HashCombine(ret, val.num_args);
    ret = dmlc::HashCombine(ret, val.subscripts);
    ret = dmlc::HashCombine(ret, val.optimize);
    return ret;
  }
};
}  // namespace std

namespace mxnet {
namespace op {

template<typename ComputeType,
         typename IntType, int kMaxNumModes_>
struct Einsum
{
    Einsum(const std::string &equation,
           const mxnet::TShape &A_shape,
           const mxnet::TShape &B_shape) :
        numModesA_(A_shape.ndim()),
        numModesB_(B_shape.ndim()),
        numModesC_(0),
        isInitialized_(false)
    {
        const auto arrow_pos = equation.find("->");
        const auto comma_pos = equation.find(",");
        const auto dots = equation.find("...");
        const bool isBroadcast = (dots != std::string::npos);
        const bool isImplicit = (arrow_pos == std::string::npos);
        if (isBroadcast) // TODO
        {
            return;
        }
        const bool usesB = (comma_pos != std::string::npos);

        size_t a_start = 0;
        size_t a_end = isImplicit ? ((comma_pos == std::string::npos) ? equation.size() : comma_pos) : 
                                    ((comma_pos == std::string::npos) ? arrow_pos : comma_pos);
        size_t b_start = usesB ? comma_pos + 1 : 0;
        size_t b_end   = usesB ? (isImplicit ? equation.size() : arrow_pos) : 0;
        size_t c_start = isImplicit ? equation.size() : arrow_pos + 2;
        size_t c_end = equation.size();


        char modeA[kMaxNumModes_ + 2];
        uint32_t numModesA = 0;
        for (int i = a_start; i < a_end && numModesA < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeA[numModesA++] = equation.at(i);
            }
        }

        char modeB[kMaxNumModes_ + 2];
        uint32_t numModesB = 0;
        for (int i = b_start; i < b_end && numModesB < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeB[numModesB++] = equation.at(i);
            }
        }

        char modeC[kMaxNumModes_ + 2];
        uint32_t numModesC = 0;
        for (int i = c_start; i < c_end && numModesC < kMaxNumModes_ + 2; ++i){
            if (equation.at(i) != ' ') // skip spaces
            {
                modeC[numModesC++] = equation.at(i);
            }
        }

        if ((numModesA != numModesA_) || (numModesB != numModesB_))
        {
            // substring size and shape don't match
            return;
        }
        if (numModesA_ > kMaxNumModes_ || numModesB_ > kMaxNumModes_)
        {
            // too many modes
            return;
        }

        /**
         * Copy all modes from modeA to modeC if they don't appear in modeB
         */
        auto copyModesIf = [](const char* modeA, uint32_t numModesA,
                const char* modeB, uint32_t numModesB,
                char* modeC, uint32_t &numModesC)
        {
            for (uint32_t i = 0; i < numModesA; i++)
            {
                auto mode = modeA[i];
                bool found = false;
                for(uint32_t j=0; j < numModesB; ++j){
                    if(mode == modeB[j])
                    {
                        found = true;
                        break;
                    }
                }

                if (!found) // is non-contracted mode
                {
                    modeC[numModesC++] = mode;
                    if (numModesC > kMaxNumModes_)
                    {
                        // too many modes
                        return false;
                    }
                }
            }
            return true;
        };


        std::array<char, kMaxNumModes_+1> implicitModeC;
        char* redirectModeC;
        if (isImplicit)
        {
            // we have to copy all non-contracted modes from A over to C
            if (copyModesIf(modeA, numModesA_, modeB, numModesB_, implicitModeC.data(), numModesC_) == false)
            {
                return;
            }
            // we have to copy all non-contracted modes from B over to C
            if (copyModesIf(modeB, numModesB_, modeA, numModesA_, implicitModeC.data(), numModesC_) == false)
            {
                return;
            }
            std::sort(implicitModeC.begin(), std::next(implicitModeC.begin(), numModesC_)); // modes are sorted w.r.t. lexical order
            implicitModeC[numModesC_] = '\0';
            redirectModeC = implicitModeC.data();
        }
        else
        {
            redirectModeC = modeC;
            numModesC_ = numModesC;
        }

        for (uint32_t i = 0; i < numModesA_; i++)
        {
            modesA_[i] = modeA[numModesA_ - i - 1];
            extentA_[i] = A_shape[numModesA_ - i - 1];
        }

        for (uint32_t i = 0; i < numModesB_; i++)
        {
            modesB_[i] = modeB[numModesB_ - i - 1];
            extentB_[i] = B_shape[numModesB_ - i - 1];
        }

        for (uint32_t i = 0; i < numModesC_; i++)
        {
            const auto mode = redirectModeC[numModesC_ - i - 1];
            modesC_[i] = mode;
            bool found = false;
            for (uint32_t j=0; j < numModesA_; ++j)
            {
                if (modesA_[j] == mode)
                {
                    extentC_[i] = extentA_[j];
                    found = true;
                    break;
                }
            }
            for (uint32_t j=0; !found && j < numModesB_; ++j)
            {
                if (modesB_[j] == mode)
                {
                    extentC_[i] = extentB_[j];
                    break;
                }
            }
        }

        isInitialized_ = true;
    }

    std::vector<IntType> getOutputShape() const
    {
        if (!isInitialized_) return {};
        std::vector<IntType> extentC(numModesC_);
        for (int i=0; i < numModesC_; ++i)
        {
            extentC[i] = extentC_.at(numModesC_ - i - 1);
        }

        return extentC;
    }

    bool isInitialized() const { return isInitialized_; }

    const int64_t* getExtentsA() const { return extentA_.data(); }
    const int64_t* getExtentsB() const { return extentB_.data(); }
    const int64_t* getExtentsC() const { return extentC_.data(); }

    const int* getModesA() const { return modesA_.data(); }
    const int* getModesB() const { return modesB_.data(); }
    const int* getModesC() const { return modesC_.data(); }

    private:
    uint32_t numModesA_;
    uint32_t numModesB_;
    uint32_t numModesC_;
    bool isInitialized_;
    std::array<int, kMaxNumModes_> modesA_;
    std::array<int, kMaxNumModes_> modesB_;
    std::array<int, kMaxNumModes_> modesC_;
    std::array<int64_t, kMaxNumModes_> extentA_;
    std::array<int64_t, kMaxNumModes_> extentB_;
    std::array<int64_t, kMaxNumModes_> extentC_;
};

/*!
 * \brief The Operator used to perform einsum using cuTensor library.
 */
template<typename DType>
class CuTensorEinsum {
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  static_assert(CUTENSOR_MAJOR >= 1 && CUTENSOR_MINOR >= 2 && CUTENSOR_PATCH >= 0, "minimal cuTENSOR 1.2.0 is required.");
 public:
  CuTensorEinsum() {
  }
  ~CuTensorEinsum() {
  }

  size_t Init(std::string equation,
              const mxnet::ShapeVector& in_shape,
              const mxnet::ShapeVector& out_shape,
              const OpContext& ctx,
              bool req_write,
              size_t prev_workspace_size,
              size_t alignment) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    CHECK_EQ(in_shape.size(), 2);
    CHECK_EQ(out_shape.size(), 1);

    constexpr hipDataType cudaType = CuTensorTypeTraits<DType>::cudaType;
    constexpr hiptensorComputeDescriptor_t cutensorType = CuTensorTypeTraits<DType>::cutensorType;
    constexpr hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;

    Einsum<DType, int, kMaxTensorRank> myEinsum(equation, in_shape[0], in_shape[1]);
    if (!myEinsum.isInitialized()) {
        CUTENSOR_CALL(HIPTENSOR_STATUS_NOT_SUPPORTED);
    }

    hiptensorTensorDescriptor_t descriptor_a;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_a,
                                               in_shape[0].ndim(),
                                               myEinsum.getExtentsA(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));
    hiptensorTensorDescriptor_t descriptor_b;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_b,
                                               in_shape[1].ndim(),
                                               myEinsum.getExtentsB(),
                                               NULL, //stride
                                               cudaType, HIPTENSOR_OP_IDENTITY));
    hiptensorTensorDescriptor_t descriptor_c;
    CUTENSOR_CALL(hiptensorInitTensorDescriptor(&s->cutensor_handle_,
                                               &descriptor_c,
                                               out_shape[0].ndim(),
                                               myEinsum.getExtentsC(),
                                               NULL, //stride
                                               cudaType,
                                               HIPTENSOR_OP_IDENTITY));

    CUTENSOR_CALL(cutensorInitContractionDescriptor(
                  &s->cutensor_handle_,
                  &descriptor_contraction,
                  &descriptor_a, myEinsum.getModesA(), alignment,
                  &descriptor_b, myEinsum.getModesB(), alignment,
                  &descriptor_c, myEinsum.getModesC(), alignment,
                  &descriptor_c, myEinsum.getModesC(), alignment,
                  cutensorType));

    CUTENSOR_CALL(cutensorInitContractionFind(&s->cutensor_handle_,
                                              &find, algo));

    if (s->cutensor_cachelines_ != nullptr)
    {
        const hiptensorAutotuneMode_t autotuneMode = HIPTENSOR_AUTOTUNE_MODE_INCREMENTAL;
        CUTENSOR_CALL(cutensorContractionFindSetAttribute(
                    &s->cutensor_handle_,
                    &find,
                    CUTENSOR_CONTRACTION_FIND_AUTOTUNE_MODE,
                    &autotuneMode,
                    sizeof(hiptensorAutotuneMode_t)));

        const uint32_t incCount = 5;
        CUTENSOR_CALL(cutensorContractionFindSetAttribute(
                    &s->cutensor_handle_,
                    &find,
                    CUTENSOR_CONTRACTION_FIND_INCREMENTAL_COUNT,
                    &incCount,
                    sizeof(uint32_t)));
    } 

    previous_workspace_size = prev_workspace_size * sizeof(DType);
    CUTENSOR_CALL(cutensorContractionGetWorkspace(&s->cutensor_handle_,
                                                  &descriptor_contraction,
                                                  &find,
                                                  HIPTENSOR_WORKSPACE_MAX,
                                                  &my_workspace_size));
    if (s->cutensor_cachelines_ == nullptr){
        CUTENSOR_CALL(cutensorInitContractionPlan(&s->cutensor_handle_,
                                              &plan,
                                              &descriptor_contraction,
                                              &find,
                                              my_workspace_size));
    }

    return my_workspace_size;
  }

  void Compute(const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               bool req_write,
               const std::vector<TBlob> &outputs,
               char* workspace) {
    Stream<gpu>* s = ctx.get_stream<gpu>();

    if (s->cutensor_cachelines_ != nullptr)
    {
        CUTENSOR_CALL(cutensorInitContractionPlan(&s->cutensor_handle_,
                                              &plan,
                                              &descriptor_contraction,
                                              &find,
                                              my_workspace_size));
    }

    const TBlob &tensor_a = inputs[0];
    const TBlob &tensor_b = inputs[1];
    const TBlob &tensor_c = outputs[0];
    const DType* tensor_a_ptr =  tensor_a.FlatTo2D<gpu, DType>(s).dptr_;
    const DType* tensor_b_ptr =  tensor_b.FlatTo2D<gpu, DType>(s).dptr_;
    DType* tensor_c_ptr =  tensor_c.FlatTo2D<gpu, DType>(s).dptr_;
    char* my_workspace(&workspace[previous_workspace_size]);
    CUTENSOR_CALL(hiptensorContraction(&s->cutensor_handle_,
                                      &plan,
                                      (void*) &alpha, tensor_a_ptr, tensor_b_ptr,
                                      (void*) &beta,  tensor_c_ptr, tensor_c_ptr,
                                      my_workspace,
                                      my_workspace_size,
                                      s->stream_));
  }

  static const uint32_t kMaxTensorRank = 12; // maximal tensor rank that is supported by cuTENSOR

  cutensorContractionDescriptor_t descriptor_contraction; // encodes the strucutre of the contraction
  hiptensorContractionPlan_t plan; // encodes the execution plan
  cutensorContractionFind_t find; // limits the search space (of viable candidates/implementations)

  // workspace
  size_t previous_workspace_size = 0;
  size_t my_workspace_size = 0;
  size_t total_workspace_size = 0;
  
  typename CuTensorTypeTraits<DType>::ScalarType alpha = 1;
  typename CuTensorTypeTraits<DType>::ScalarType beta = 0;
};
// end CuTensorEinsum class

template<typename DType>
class EinsumOpGPU {

 public:
  EinsumOpGPU() {
  }
  ~EinsumOpGPU() {
  }

  void InitCuTensorGrad(std::string equation,
                        const mxnet::ShapeVector& in_shape,
                        const mxnet::ShapeVector& out_shape,
                        const OpContext &ctx,
                        size_t &pos_cutensor_op,
                        size_t temp_grad_size_aligned){
    bool req_write = true;
    int comma_pos = equation.find(",");
    int arrow_pos = equation.find("->", comma_pos + 1);
    int len_op2 = arrow_pos - comma_pos - 1;

    // gradient for first operand
    mxnet::ShapeVector grad_op1_input_shapes;
    mxnet::ShapeVector grad_op1_output_shapes;
    grad_op1_input_shapes.push_back(in_shape[0]);
    grad_op1_input_shapes.push_back(in_shape[2]);
    grad_op1_output_shapes.push_back(out_shape[0]);
    std::string grad_operand1_equation = equation.substr(arrow_pos + 2);
    grad_operand1_equation += ",";
    grad_operand1_equation += equation.substr(comma_pos + 1, len_op2);
    grad_operand1_equation += "->";
    grad_operand1_equation += equation.substr(0, comma_pos);
    bwd_cutensor_ops.push_back(CuTensorEinsum<DType>());
    size_t req_workspace =
      bwd_cutensor_ops[pos_cutensor_op].Init(grad_operand1_equation,
                                             grad_op1_input_shapes,
                                             grad_op1_output_shapes,
                                             ctx, req_write,
                                             temp_grad_size_aligned,
                                             dptr_alignment);
    if (req_workspace > max_workspace_cutensor) max_workspace_cutensor = req_workspace;
    pos_cutensor_op++;

    // gradient for second operand
    mxnet::ShapeVector grad_op2_input_shapes;
    mxnet::ShapeVector grad_op2_output_shapes;
    grad_op2_input_shapes.push_back(in_shape[1]);
    grad_op2_input_shapes.push_back(in_shape[0]);
    grad_op2_output_shapes.push_back(out_shape[1]);
    std::string grad_operand2_equation = equation.substr(0, comma_pos);
    grad_operand2_equation += ",";
    grad_operand2_equation += equation.substr(arrow_pos + 2);
    grad_operand2_equation += "->";
    grad_operand2_equation += equation.substr(comma_pos + 1, len_op2);
    bwd_cutensor_ops.push_back(CuTensorEinsum<DType>());
    req_workspace =
      bwd_cutensor_ops[pos_cutensor_op].Init(grad_operand2_equation,
                                             grad_op2_input_shapes,
                                             grad_op2_output_shapes,
                                             ctx, req_write,
                                             temp_grad_size_aligned,
                                             dptr_alignment);
    if (req_workspace > max_workspace_cutensor) max_workspace_cutensor = req_workspace;
    pos_cutensor_op++;
  }

  void Init(const EinsumOp& state,
            const mxnet::ShapeVector& in_shape,
            const mxnet::ShapeVector& out_shape,
            const OpContext& ctx,
            bool req_write,
            bool is_backward) {
    if (!is_backward) {
      // forward
      if (state.num_args == 2) {
        fwd_cutensor_ops.push_back(CuTensorEinsum<DType>());
        max_workspace_cutensor = fwd_cutensor_ops[0].Init(state.subscripts,
                                                          in_shape, out_shape,
                                                          ctx, req_write,
                                                          0, dptr_alignment);
      } else {
        // more than 2 operands, optimal path
        paths_len = state.paths.size();
        max_workspace_cutensor = 0;
        mxnet::ShapeVector operands_shape(in_shape);
        for (int i = 0; i < paths_len; ++i) {
          bool handle_out = (i == paths_len - 1);
          mxnet::ShapeVector tmp_in_shape;
          mxnet::ShapeVector tmp_out_shape;
          // remove inds from right to left
          for (const int& p : state.paths[i].contract_inds) {
            tmp_in_shape.push_back(operands_shape[p]);
            operands_shape.erase(operands_shape.begin() + p);
          }
          if (handle_out) tmp_out_shape.push_back(out_shape[0]);
          else tmp_out_shape.push_back(state.paths[i].oshape);
          fwd_cutensor_ops.push_back(CuTensorEinsum<DType>());

          size_t req_workspace = fwd_cutensor_ops[i].Init(state.paths[i].einsum_str,
                                                          tmp_in_shape,
                                                          tmp_out_shape,
                                                          ctx, req_write,
                                                          0, dptr_alignment);
          if (req_workspace > max_workspace_cutensor) max_workspace_cutensor = req_workspace;
          temp_ouputs_size += state.paths[i].oshape.Size();
          if (!handle_out) {
            operands_shape.push_back(state.paths[i].oshape);
          }
        }
      }
    } else {
      // backward
      max_workspace_cutensor = 0;
      size_t pos_cutensor_bwd_op = 0;
      if (state.num_args == 2) {
        InitCuTensorGrad(state.subscripts,
                         in_shape, out_shape,
                         ctx, pos_cutensor_bwd_op, 0);
        total_workspace = max_workspace_cutensor/sizeof(DType);
      } else {
        // more than 2 operands, optimal path
        paths_len = state.paths.size();
        // replay the forward process
        bwd_op_idx.resize(paths_len + 1);
        for (int i = 0; i <= paths_len; ++i) {
          if (i == 0) {
            bwd_op_idx[i].reserve(state.num_args);
            for (int j = 0; j < state.num_args; ++j) {
              bwd_op_idx[i].push_back(j + 1);
            }
          } else {
            bwd_op_idx[i] = bwd_op_idx[i - 1];
            // remove inds from right to left
            for (const int& p : state.paths[i - 1].contract_inds) {
              bwd_op_idx[i].erase(bwd_op_idx[i].begin() + p);
            }
            bwd_op_idx[i].push_back(-static_cast<int>(i - 1));
          }
        }
        // calculate amount mem for temporal grads
        for (int i = 0; i + 1 < paths_len; ++i) {
          temp_grads_size += state.paths[i].oshape.Size();
        }
        temp_grads_size_aligned = RoundToMultiple(temp_grads_size, dptr_alignment);
        // go through the paths in the reversed order
        mxnet::ShapeVector temp_in_shape, temp_out_shape;
        for (int i = paths_len - 1; i >= 0; i--) {
          temp_in_shape.clear();
          temp_out_shape.clear();
          bool handle_out = (i == paths_len - 1);
          if (handle_out) {
            // grad_out
            temp_in_shape.push_back(in_shape[0]);
          } else {
            temp_in_shape.push_back(state.paths[i].oshape);
          }
          for (auto p : state.paths[i].contract_inds) {
            int idx = bwd_op_idx[i][p];
            if (idx >= 1) {
              temp_in_shape.push_back(in_shape[idx]);
              temp_out_shape.push_back(out_shape[idx - 1]);
            } else {
              temp_in_shape.push_back(state.paths[-idx].oshape);
              temp_out_shape.push_back(state.paths[-idx].oshape);
            }
          }
          CHECK_EQ(temp_in_shape.size(), 3U);
          CHECK_EQ(temp_out_shape.size(), 2U);

          InitCuTensorGrad(state.paths[i].einsum_str,
                           temp_in_shape, temp_out_shape,
                           ctx, pos_cutensor_bwd_op,
                           temp_grads_size_aligned);
        }
        total_workspace = max_workspace_cutensor/sizeof(DType) +
                          temp_grads_size;
      }
    }
  }

  void Forward(const EinsumOp& state,
               const OpContext &ctx,
               const std::vector<TBlob> &inputs,
               const std::vector<OpReqType>& req,
               const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    bool req_write = false;
    // cuTensor workspace
    Tensor<gpu, 1, char> cutensor_workspace =
        ctx.requested[0].get_space_typed<gpu, 1, char>(Shape1(max_workspace_cutensor), s);

    if (state.num_args == 2) {
      fwd_cutensor_ops[0].Compute(ctx, inputs, req_write, outputs, cutensor_workspace.dptr_);
    } else {
      // more than 2 operands, compute optimal path
      std::vector<TBlob> operands(inputs);
      std::vector<TBlob> tmp_operands;

      // temporal space shared with backward: stateful
      std::vector<TBlob> temp_space_vec(paths_len - 1);
      Tensor<gpu, 1, DType> temp_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      size_t begin = 0;
      for (int i = 0; i < paths_len - 1; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_space_vec[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      for (int i = 0; i < paths_len; ++i) {
        bool handle_out = (i == paths_len - 1);
        tmp_operands.clear();
        // remove inds from right to left
        for (const int& p : state.paths[i].contract_inds) {
          tmp_operands.push_back(operands[p]);
          operands.erase(operands.begin() + p);
        }
        fwd_cutensor_ops[i].Compute(ctx, tmp_operands, req_write,
                                    handle_out ? outputs :
                                                 std::vector<TBlob>{temp_space_vec[i]},
                                    cutensor_workspace.dptr_);
        if (!handle_out) {
          operands.push_back(temp_space_vec[i]);
        }
      }
    }
  }

  void ComputeGradients(const std::string &equation,
                        const std::vector<TBlob> &inputs,
                        const std::vector<TBlob> &outputs,
                        const OpContext &ctx,
                        size_t &pos_cutensor_op,
                        Tensor<gpu, 1, DType> &workspace){
    char* workspace_ptr = reinterpret_cast<char*>(workspace.dptr_);
    bool req_write = true;
    // gradient for first operand
    std::vector<TBlob> grad_operand1_inputs;
    std::vector<TBlob> grad_operand1_outputs;
    grad_operand1_inputs.push_back(inputs[0]);
    grad_operand1_inputs.push_back(inputs[2]);
    grad_operand1_outputs.push_back(outputs[0]);
    bwd_cutensor_ops[pos_cutensor_op].Compute(ctx, 
                                              grad_operand1_inputs, 
                                              req_write,
                                              grad_operand1_outputs,
                                              workspace_ptr);
    pos_cutensor_op ++;
    // gradient for second operand
    std::vector<TBlob> grad_operand2_inputs;
    std::vector<TBlob> grad_operand2_outputs;
    grad_operand2_inputs.push_back(inputs[1]);
    grad_operand2_inputs.push_back(inputs[0]);
    grad_operand2_outputs.push_back(outputs[1]);
    bwd_cutensor_ops[pos_cutensor_op].Compute(ctx, 
                                              grad_operand2_inputs, 
                                              req_write,
                                              grad_operand2_outputs,
                                              workspace_ptr);
    pos_cutensor_op++;
  }

  void Backward(const EinsumOp& state,
                const OpContext &ctx,
                const std::vector<TBlob> &inputs,
                const std::vector<OpReqType>& req,
                const std::vector<TBlob>& outputs) {
    mshadow::Stream<gpu> *s = ctx.get_stream<gpu>();
    auto req_write = req[0] == kWriteTo;
    size_t pos_cutensor_op = 0;
    if (state.num_args == 2) {
      // workspace for cuTensor
      Tensor<gpu, 1, DType> temp_space =
        ctx.requested[0].get_space_typed<gpu, 1, DType>(Shape1(total_workspace), s);
      // inputs: out_grad, operand1, operand2
      // outputs: grad_operand1, grad_operand2
      ComputeGradients(state.subscripts,
                       inputs, outputs,
                       ctx, pos_cutensor_op,
                       temp_space);
    } else {
      // more than 2 operands, compute optimal path
      // outputs from forward pass, no need to be re-computed, take from state
      Tensor<gpu, 1, DType> ndarray_space = state.tempspace->data().FlatTo1D<gpu, DType>();
      std::vector<TBlob> temp_data(paths_len - 1);
      size_t begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(ndarray_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_data[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // workspace (temporal grad + cuTensor)
      std::vector<TBlob> temp_grad(paths_len - 1);
      Tensor<gpu, 1, DType> temp_space =
        ctx.requested[0].get_space_typed<gpu, 1, DType>(Shape1(total_workspace), s);
      begin = 0;
      for (int i = 0; i + 1 < paths_len; ++i) {
        TBlob tblob = TBlob(temp_space.Slice(begin, begin + state.paths[i].oshape.Size()));
        temp_grad[i] = tblob.reshape(state.paths[i].oshape);
        begin = begin + state.paths[i].oshape.Size();
      }
      // go through the paths in the reversed order
      std::vector<TBlob> temp_inputs, temp_outputs;
      //std::vector<OpReqType> temp_req;
      for (int i = paths_len - 1; i >= 0; i--) {
        temp_inputs.clear();
        temp_outputs.clear();
        //temp_req.clear();
        bool handle_out = (i == paths_len - 1);
        if (handle_out) {
          // grad_out
          temp_inputs.push_back(inputs[0]);
        } else {
          temp_inputs.push_back(temp_grad[i]);
        }
        for (auto p : state.paths[i].contract_inds) {
          int idx = bwd_op_idx[i][p];
          if (idx >= 1) {
            temp_inputs.push_back(inputs[idx]);
            temp_outputs.push_back(outputs[idx - 1]);
            //temp_req.push_back(req[idx - 1]);
          } else {
            temp_inputs.push_back(temp_data[-idx]);
            temp_outputs.push_back(temp_grad[-idx]);
            //temp_req.push_back(OpReqType::kWriteTo);
          }
        }
        CHECK_EQ(temp_inputs.size(), 3U);
        CHECK_EQ(temp_outputs.size(), 2U);
        //CHECK_EQ(temp_req.size(), 2U);

        ComputeGradients(state.paths[i].einsum_str,
                         temp_inputs, temp_outputs,
                         ctx, pos_cutensor_op,
                         temp_space);
      }
    }
  }

  int paths_len = 0;
  // cutensor ops for the forward and backward passs:
  std::vector<CuTensorEinsum<DType>> fwd_cutensor_ops;
  std::vector<CuTensorEinsum<DType>> bwd_cutensor_ops;
  std::vector<std::vector<int> > bwd_op_idx;
  
  const size_t dptr_alignment = 512;
  size_t temp_ouputs_size = 0; 
  size_t temp_grads_size = 0;
  size_t temp_grads_size_aligned = 0;
  size_t max_workspace_cutensor = 0;
  size_t total_workspace = 0;
};

typedef ParamOpSign<EinsumOp> EinsumSignature;
template<typename DType>
static EinsumOpGPU<DType>& GetEinsumOpGPU(const EinsumOp& state,
                                          const mxnet::ShapeVector& in_shape,
                                          const mxnet::ShapeVector& out_shape,
                                          const OpContext& ctx,
                                          bool req_write,
                                          bool is_backward) {
#if DMLC_CXX11_THREAD_LOCAL
  static thread_local std::unordered_map<EinsumSignature,
                                         std::shared_ptr<EinsumOpGPU<DType> >,
                                         OpHash> ops;
#else
  static MX_THREAD_LOCAL std::unordered_map<EinsumSignature,
                                            std::shared_ptr<EinsumOpGPU<DType> >,
                                            OpHash> ops;
#endif
  EinsumSignature key(state);
  size_t ndim = 0;
  for (auto &s : in_shape)
    ndim += s.ndim();
  for (auto &s : out_shape)
    ndim += s.ndim();
  key.Reserve(ndim + // for in and out shapes
              1 + // for dev_id
              1 + // for req_write
              1 ); // is_backward
  key.AddSign(in_shape);
  key.AddSign(out_shape);
  key.AddSign(ctx.run_ctx.ctx.dev_id);
  key.AddSign(req_write ? 1 : 0);
  key.AddSign(is_backward ? 1 : 0);

  auto it = ops.find(key);
  if (it == ops.end()) {
    std::shared_ptr<EinsumOpGPU<DType>> op(new EinsumOpGPU<DType>());
    auto ins_ret = ops.insert(std::pair<EinsumSignature, std::shared_ptr<EinsumOpGPU<DType>>>(
                              key, op));
    CHECK(ins_ret.second);
    it = ins_ret.first;
    it->second->Init(state,
                     in_shape, out_shape,
                     ctx, req_write, is_backward);
  }
  return *it->second;
}
#endif

inline void NumpyEinsumForwardGpu(const OpStatePtr& state_ptr,
                                  const OpContext& ctx,
                                  const std::vector<TBlob>& inputs,
                                  const std::vector<OpReqType>& req,
                                  const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = false;
  if (state.num_args <= 1) {
    NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    if (state.num_args > 2) {
      std::vector<Step>& paths = state.paths;
      std::vector<std::vector<int> > pos;
      std::string string_repr;
      paths = einsum_path(state.subscripts, inputs, true, ctx.run_ctx, &pos, &string_repr);
    }
    mxnet::ShapeVector in_shape(inputs.size());
    mxnet::ShapeVector out_shape(1, outputs[0].shape_);
    for (size_t i = 0; i < in_shape.size(); i++)
      in_shape[i] = inputs[i].shape_;

    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, in_shape, out_shape,
           ctx, req_write, false);
      state.tempspace.reset<NDArray>(new NDArray(TShape(Shape1(op.temp_ouputs_size)),
                                               ctx.run_ctx.ctx,
                                               false,
                                               outputs[0].type_flag_));
      op.Forward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumForward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

inline void NumpyEinsumBackwardGpu(const OpStatePtr& state_ptr,
                                   const OpContext& ctx,
                                   const std::vector<TBlob>& inputs,
                                   const std::vector<OpReqType>& req,
                                   const std::vector<TBlob>& outputs) {
#if MXNET_USE_CUTENSOR == 1
  // cutensor only available for compute capability larger or equal to 6.0
  STATIC_ASSERT_CUDNN_VERSION_GE(6000);
  const EinsumOp& state = state_ptr.get_state<EinsumOp>();
  auto req_write = req[0] == kWriteTo;
  if (state.num_args <= 1) {
    NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
  } else {
    mxnet::ShapeVector in_shape(inputs.size());
    mxnet::ShapeVector out_shape(outputs.size());
    for (size_t i = 0; i < in_shape.size(); i++)
      in_shape[i] = inputs[i].shape_;
    for (size_t i = 0; i < out_shape.size(); i++)
      out_shape[i] = outputs[i].shape_;
    MSHADOW_REAL_TYPE_SWITCH(outputs[0].type_flag_, DType, {
      EinsumOpGPU<DType> &op = GetEinsumOpGPU<DType>
          (state, in_shape, out_shape,
           ctx, req_write, true);
      op.Backward(state, ctx, inputs, req, outputs);
    });
  }
#else
  NumpyEinsumBackward<gpu>(state_ptr, ctx, inputs, req, outputs);
#endif
}

NNVM_REGISTER_OP(_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumForwardGpu);

NNVM_REGISTER_OP(_backward_npi_einsum)
.set_attr<FStatefulCompute>("FStatefulCompute<gpu>", NumpyEinsumBackwardGpu);

}  // namespace op
}  // namespace mxnet
